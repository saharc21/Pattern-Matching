#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <pthread.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "slist.c"
#include "pattern_matching_aho.c"


__device__ void print_pm(pm_t *pm);
__device__ void print_state(pm_state_t *state, int tabs, int *is_need, int is_get);
__device__ void print_tabs(int tabs, int *is_need);
__global__ void search_patterns(pm_t *pm,  char *s,int lenOfS, int offset);
__global__ void initilaizeTree_aho(pm_t * cu_pm);
void cleanup_destroy(slist_t *list);

char s1[PM_CHARACTERS];
__device__  char * cu_s1;
__device__  pm_t * cu_pm; 
int MAX_THEADS_FOR_EPOCH;
int NUM_OF_THREADS;


int main(int argc, char *argv[]) // ********************* Aho - parallel ***********************
{
    FILE *fp;
    fp = fopen("/home/jceproject/Desktop/Final Project/Pattern-Matching/chars_stream.txt", "r");
    fgets(s1, PM_CHARACTERS+1, (FILE *)fp);
    int numOfChars = strlen(strtok(s1, "\0"));
    
    MAX_THEADS_FOR_EPOCH = 1024;
    
    if(numOfChars <= MAX_THEADS_FOR_EPOCH){    
        NUM_OF_THREADS = numOfChars;
    }    

    else{
        NUM_OF_THREADS = MAX_THEADS_FOR_EPOCH;
    }
    

    // pm_t * cu_pm;
    hipMallocManaged(&cu_pm, sizeof(pm_t));
    

    if (!cu_pm)
    {
        return -1;
    }
    
    // if (pm_init(pm) == -1)
    // {
    //     free(pm);
    //     return -1;
    // }

    initilaizeTree_aho<<<1,1>>>(cu_pm);
    // char * cu_s1;
    hipMallocManaged(&cu_s1, (numOfChars+1)*sizeof(char));
    hipMemcpy(cu_s1, s1,numOfChars,hipMemcpyHostToDevice);

    
    clock_t begin;
    clock_t end;
    begin = clock();

    int len = strlen(s1);


    // printf("gpu tree: \n");
    // print_pm(cu_pm);

    int loop_limit = numOfChars/NUM_OF_THREADS;
    if(numOfChars%NUM_OF_THREADS > 0)
    {
        loop_limit++;
    }    
    int offset;
    int threads = NUM_OF_THREADS;
    for (offset = 0; offset <= loop_limit; offset++){
        if(offset == loop_limit){
            threads = numOfChars%NUM_OF_THREADS;
        }
        search_patterns<<<threads,1>>>(&(*cu_pm), cu_s1, len, MAX_THEADS_FOR_EPOCH*offset);
    }
    hipDeviceSynchronize();


    // hipFree(cu_pm);
    // hipFree(cu_s1);
    end = clock();
    printf("\nExecuted time is: %f ms. \n\n", ((double)(end - begin) / CLOCKS_PER_SEC) * 1000);
    
    // cleanup_destroy(list);

    end = clock();
    // fclose(fp);
    return 0;
}


__global__ void initilaizeTree_aho(pm_t * cu_pm)
{
    if(pm_init_gpu(cu_pm) == -1)
    {
        return;
    }
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"DR5HY@yLcG*6GD7sXf1paMwfLiD&Gdub98QW0FzqpWjEHxMQZ0kJoZo#hJVJ9QGD3VdtIadTp4me*sfAzT3NjL0$4L2FC6NzmDd6mhtyW04o@*qO#le31L^MO$X#CpFYxHSegmPDjiCf8R^cmjtARILA*Z^e!n&pPEC&WKKNHO7I4P6XPA0&#75RJ7ghWsHXz^r3vSh@OzJK#jqaZBvz9x4EF*$hlfcrykrKdLgio7AaxLiuDD805i@4&s4WWusG7fsf5iNllmMrVkb7RtzSU@fZo1bLVaceLA#CkuWSpvaD0b#ZGwkcTesehS5UwKUObzILks#sdUJDBdadWpgP&lanjk^CbqjbumPvz2chvCa&Sc120z5p60TEDBc9esY3vbtq8w5j^M0K$$UYZB2NMAUjJ*kK^1SoX5f03yxAh0#!R$92Ui&88zhwxsnq2evaygzksj##w1Sx^jvknA&Vyba^Dt1j8CM", 479) == -1){
        return;
    }
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"#^1wds5KDaplo&!mEFt7Xak&SFpa0fNP8xZwdmRXS7tZ2!B9P4na&yHXT*JAsuGUmeYypacsvkxnkUDgC*GWmUBf5WKT8E2k!UF@pK3g86ansmrmOfspzeSGjJmjfmg9#*pOYs@b^cGZLycrtv@*wrxCgeVvb&4v3mZwRwDgdEPZMQX0lyANM23HL3oWm#dm2nS6A!OAGq$QjoA9*DK&*ffQ2&wZxJf06Vxck1om!ccKe7qpqZk5iJECurt!3s6EpTPdpgxeKAZLlR4Bt1V#5ZBfX4tW0^3I0zlJvomrNOiOKi!vK^8lX36JkuJCzhrvAKk#aC^RtQAtS$qK^A*30I^$Ed3KZ82rgwG7z4mswABIzOH*3bm!6C@dCx", 378) == -1){
        return;
    }
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"lKEX*VA$uFqJ2FfJE5*EI!MU5vMg!KGq9xP$fAshHrIVXgQfL45&1tT&TOUcTcKUYGoAgTEIrl6NiL8n&DPIvmomUpBp2sHM#*g1V2&xsYFOOznYQwzoqnBp2w*P", 124) == -1){
        return;
    }    
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"dA1sI0NGxYzGPWQreWrptknA9SMf44Bfl9hXDAV!TvWSGB!1e66$2wG6tx&Re*LoEBwJ!dUL@qOJuJ2kffeazN1Mhcvuc4$J7Xjox$1fAwSW6ziIvlv7PstENidAXl7DnlR4W7*yCjz$XhPvmWLflmvohg!XxTI*c#Ra5QRL2HKmc&PCzjQM*wzjSbbxLqC*G$Qe^nFIM8nEzfV4keXikucg$Z1GCW04eBYTWp*HCwrHHGqtC*XmRsPx@iGMQg5yGkeXQMiiQJUyLnp1f3PC4evR@^zijZ$H&XkZ7XuTUTcVV8vkIUpzoTRjYw$C1arvd&Y!&rFY35*RFDj!bv&&yl*sN4W3ALkDGXudUKRlPFYy!D89kXQmwkyHjO*&RAv#eJ2vBN7#*NMN*eY0bD$xQ0Rn7ywTUcch^fiU@3s$$2oGH2zjVLKXhHFVeRb7X9al9T^vwldPKFJDgqJgVky!QgWQ5g#*^lsEMykUV6o#h2UPhoy3@rqscqeBJ!GJwEJWj$Li^bynEWhtTsojwCiaB29gU&thp0O5$PI1Lq9UzpLTc8ZZ1$OgJSN11xJzpjC1&mNh7T@wD8j&M&Rjk^TXZckBHT9fVmS1!F0GHslpaBTZZ#DD6p@!CkDk", 632) == -1){
        return;
    }
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"gGoojjNNZnj09JmxraEvWyaO9raPqdABd0iZSqoMVyPtyIfpLolELRVtyAjYyIE4x64spFYW6jXCE5CIRMdxaO0YfzlvkFYHwM1gSUbOqxuXkIYg9WFuQ6MG!VFoyoJ1xk0Y96mSP7eUhOmZeU9rzItkRzRVfvlKSvmxYwX7yoOIPVcxhL46ufZnn199V225aeKi8qXrCoUxiGELsNpV", 212) == -1){
        return;
    }
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"E4yE1Zj7vkE2mNzp8Ikimo2lIEAFFyKZEsz7N7Sue629yw8RJ3zvHxWleH9KJqt0t9GTfL75eZqayVQOzHagHwLdacJHjaZ2ul4zHCb1af7CPpsSp6wstt8xBhqAaSydUCJcoFyetcxEh0zM1MiMDKEyDHmq4OhvSMFbI3SbgGoZ8SiMvCcWX2b0YsQVgF65LoEEKIL71xye0pBNxD4bNoQgvFTcNxK33mGB37oNekERK5qUOvO9xpKlWIaTDtkOTTnXL6WjK0WKbONOCw8x1kCKwk12aVUgM27L3kLTIdo0s6VUVAAGeqYFNRYzyn2F7qJSpEiygYaNdLkiJdLcf0QguBhhIOfTKh4henpss2Z2wBStst1lZvrZL6R5OjwMEFsUEsUjS8z0nOoFEAl1TOY4P9elaAPQ7ornOFAUybAVOZEmlu2atvLJVvv1VCsIEmwnezgyULFG886hbVRO5Z9uxWti2G8FmcI09JQkuXZ3vAyyEUVmLSJhc1tKMVpv8dRKq61ROSfs6f1qhepGxoJZQTkMRMaSii71JW6Fv0XzSSrsYEylWgkRNS1JUkB3Qe8YeryJ9ubX5I5VCd0JoStAY704klkXDUE2bKNMFALcFNjVDUdjpeu7q1TslRLFbpSrdfionpmmFyIzopE4CAYkz2jlablPHLUeBMjGgiBgbYOulSIKSuwx7rlvFCbhyTsibHtZxEY6TQEOkTWSPMXqp", 729) == -1){
        return;
    }
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"YBx1TflInCLoTvSMvcgYv2Xd916G1nwiVqdpgmFdvFykLJPJRtSfVD2oiGccXNeSuTq1Pqt25hfFtIKGJDO8SVQdI5W24qLsUUJIgo31tAjbg4D6HFngMcApiewrd5CAWW41an8N5h6ZXtYM5pLytzkWddE4esqvEKHM9eRBdVpkbC0pXmVYeNCoSSjR7pwXxfSw4nk7Q0LSAdbbMgOIijc8z7wvfRLdbzuIgjVbR9USyiNm4fSYizzqc4fbQGdCx7nuNk1hxmFhS3Wpp70rP3Q7hrD4VJe3p2HlneNnpoLbJOWhJwVt7ze", 311) == -1){
        return;
    }    
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"lC14n8wwW12TxLorJKoO5PLRqSuafVm5MFC2OEzhvY5XD07nHYwGdzfDQ8XmUzUmyKNPjBp62AxJAJ97V9WPYAKHJDmKZ4AvSH1n2Ho651F9WgqbXzj3Abdr3m5wtDY8cb3kS3lRYcmNepUlOSuHw2Gy3EtPFLHVLyF2Fjls0mK6N2kxOR023jQTWwpqdHUN6MnTLn61Mx2c9sDrwSmvxaPcsGXhA6pnnPczdJhts6XvdIXmKaQzYpaC3niN82tSjwtR0VSsOeAE3WHZvjTxAduYagif23clk23QIdoenFqpxnqBLhFD9ShEfSuot28Ta0hcaIyxKuMi9KjD1f8HRM7mZ7zwQ3KaeSLoi7ovgWcUXMFkkCQsZ6rFLZlGRgBEFz2rfwXOmU0rGA94DIejZ6zy8VtI6D2nY13fsyPPGTpx80OwcCQhDEoJ83bEd1zlzwK9F15RDupRAmpy6eMz7BRosGy3rqgtauypdqbQ7vxWJ62X3dALkAwWgUu1WQNPRQpax2cusZ12igIJxRqd2MIMJw93C2BAU4eXphcFimgGLWlS3mUAtDP8ZUHqGjkACC4AQ6Fy3CigMKpLWAn7wPytFDkTXmVb0EeO0Mc6SEOOSWXbXukBT37kPMJlDcFL0pQBDqsuUp5Tny6EU", 657) == -1){
        return;
    }    
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"mjXnO5lUQKGB1RnpYMXq72alSUcZLthkcvCYuwmeTfThqjVbe0PzN6hId9a6u9DGxL37pbPcyBycDYWZmhwPF6CdQzyNxlAhLyjU0FG9jYICzxsGt", 113) == -1){
        return;
    }                
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"kDR1XpjpFrgtED8y24YD76zgJz5hIAEfRJylpD7wO8oHE0xyjPUMTcqEabuZtvnUNgfZZjclcPlwOzJALTQQ1KzlGzfrGxKnbxRYiCw0IXgfdgAINS9gaNTVVT2AYIfxG5oiTqSFGNe8mx2inlzPHJSdZLx7Gh0Rmncf5MESgPxdkVKHxS0dGHEx9WIbKiimQMy5LHl2RwRTMWybSy8X1PYiXcAGw1x9HQ7QjavVkjBBLt8GnbeulhV0dN2wTO9gnoIjhulLzRltiHx57vafQM4fOalysOUqfhUppN494uD482cWkuNq08bAaNjaNq19gL9hI3AJNXzFl7KH1f6h3xd0qGuunOQv5N25U9X676qRm3hVlkgra8EVKRMW2vbCO9SE55py9k7Ehg6lrnPy0WayZfJly2ZlgwtFqVFR7dBXYklRZcru3RQdVDFOvObNJmJPx0A82LC1Q0bqpAzuSQQ2mOWKhg6bEF0q2tWSB3d0I9m8I97MoCKDWDfIHBrpHLW1GbDJJdZCePakRSn6ityD0reUreEv3Nk3oKGY3mhVRqvwq1K374D80HBudTsYNP6xa1yJtANEus3YnqI38AQ9eu46q1xfmYfY28V9mvOviAnj3G9nGRkdIzO6CkNIPWZ7T0TTlEmftZKe1MIXZYW2AnIXaSn5YCmzT6rYqIpT0lO4TGTX8Nu2Aj8GDbhAj0Zs0cnxS5b3FSylzp8MHuJsdyNxgMGTeNtGloWQwvfhHdCsj9rEbchclaBriYZtx7rzaxbIB", 777) == -1){
        return;
    }      
 
 
    if (pm_makeFSM_gpu(cu_pm) == -1){
        return;
    }
}

__global__ void search_patterns(pm_t *pm,  char *s, int lenOfS, int offset)
{
    pm_fsm_search(pm->zerostate, (unsigned char *)(&s[offset + blockIdx.x]), lenOfS-blockIdx.x, offset + blockIdx.x);
}

void cleanup_destroy( slist_t *list ){
    slist_destroy(list, SLIST_FREE_DATA);
    free(list);
    // pm_destroy(pm);
}

__device__ void print_pm(pm_t *pm)
{
    if (!pm)
    {
        return;
    }
    printf("state(id, fail state id)\n");
    printf("(root)--------");
    int *b = (int *)malloc(sizeof(int) * 100);
    int i;
    for (i = 0; i < 100; i++)
    {
        b[i] = 0;
    }
    print_state(pm->zerostate, 1, b, 0);
    printf("\n");
    free(b);
}

__device__ void print_state(pm_state_t *state, int tabs, int *is_need, int is_get)
{
    slist_node_t *node = slist_head(state->_transitions);
    if (!node)
    {
        printf("\n");
        print_tabs(tabs, is_need);
        return;
    }
    int use_tabs = 0;
    while (node)
    {
        pm_labeled_edge_t *edge = (pm_labeled_edge_t *)slist_data(node);
        int is_out_state = slist_head(edge->state->output) ? 0 : -1;
        if (use_tabs != 0)
        {
            print_tabs(tabs, is_need);
            printf("%c---", edge->label);
        }

        else
        {
            printf("--|%c---", edge->label);
            use_tabs++;
        }

        int id = edge->state->id;
        int fail_id = edge->state->fail ? edge->state->fail->id : 0;
        is_out_state == 0 ? printf("(") : printf("-");
        printf("(");
        if (id < 10)
            printf(" ");
        printf("%d,", id);
        if (fail_id < 10)
            printf(" ");
        printf("%d)", fail_id);
        is_out_state == 0 ? printf(")") : printf("-");
        node = slist_next(node);
        if (node)
        {
            is_need[state->depth] = 1;
        }

        else
        {
            is_need[state->depth] = 0;
        }
        print_state(edge->state, tabs + 1, is_need, is_get);
    }
    hipDeviceSynchronize();
}

__device__ void print_tabs(int tabs, int *is_need)
{
    int i;
    for (i = 0; i < tabs; i++)
    {
        printf("\t\t\t\t");
        if (is_need[i] == 1)
        {
            printf("|");
        }
    }
}