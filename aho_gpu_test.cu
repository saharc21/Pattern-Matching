#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <pthread.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "slist.c"
#include "pattern_matching_aho.c"


__device__ void print_pm(pm_t *pm);
__device__ void print_state(pm_state_t *state, int tabs, int *is_need, int is_get);
__device__ void print_tabs(int tabs, int *is_need);
__global__ void search_patterns(pm_t *pm,  char *s,int lenOfS, slist_t *list, int offset);
__global__ void initilaizeTree(pm_t * cu_pm);
void cleanup_destroy(slist_t *list);

char s1[PM_CHARACTERS];
int MAX_THEADS_FOR_EPOCH;
int NUM_OF_THREADS;


int main(int argc, char *argv[]) // ********************* Aho - parallel ***********************
{
    FILE *fp;
    fp = fopen("/home/jceproject/Desktop/Final Project/Pattern-Matching/chars_test.txt", "r");
    fgets(s1, PM_CHARACTERS, (FILE *)fp);
    int numOfChars = strlen(strtok(s1, "\0"));
    
    MAX_THEADS_FOR_EPOCH = 1024;
    
    if(numOfChars< MAX_THEADS_FOR_EPOCH){    
        NUM_OF_THREADS = numOfChars;
    }    

    else{
        NUM_OF_THREADS = MAX_THEADS_FOR_EPOCH;
    }
    
    // pthread_t *tids = (pthread_t *)malloc(sizeof(pthread_t) * NUM_OF_THREADS);
    
    // pm_t *pm = (pm_t *)malloc(sizeof(pm_t));
    pm_t * cu_pm;
    hipMallocManaged(&cu_pm, sizeof(pm_t));
    
    if (!cu_pm)
    {
        return -1;
    }
    
    // if (pm_init(pm) == -1)
    // {
    //     free(pm);
    //     return -1;
    // }

    initilaizeTree<<<1,1>>>(cu_pm);
    char * cu_s1;
    hipMallocManaged(&cu_s1, (numOfChars+1)*sizeof(char));
    hipMemcpy(cu_s1, s1,numOfChars,hipMemcpyHostToDevice);

    
    clock_t begin;
    clock_t end;
    begin = clock();
    
    // hipMemcpy(cu_pm, pm,sizeof(*pm),hipMemcpyHostToDevice);

    // hipMallocManaged(&(cu_pm->zerostate), sizeof(pm->zerostate));
    // hipMemcpy(cu_pm->zerostate, pm->zerostate,sizeof(pm->zerostate),hipMemcpyHostToDevice);
    
    // hipMallocManaged(&(cu_pm->zerostate->_transitions), sizeof(pm->zerostate->_transitions));
    // hipMemcpy(cu_pm->zerostate->_transitions, pm->zerostate->_transitions,sizeof(pm->zerostate->_transitions),hipMemcpyHostToDevice);



    slist_t *list = NULL;
    int len = strlen(s1);


    // printf("gpu tree: \n");
    // print_pm(cu_pm);

    int loop_limit = (numOfChars%NUM_OF_THREADS)/MAX_THEADS_FOR_EPOCH;
    if(numOfChars%NUM_OF_THREADS > 0)
        loop_limit++;
    int offset;
    for (offset = 0; offset <= loop_limit; offset++){
        search_patterns<<< 1, NUM_OF_THREADS>>>(&(*cu_pm), cu_s1, len, list, MAX_THEADS_FOR_EPOCH*offset);
    }

    printf("in main\n");
    hipDeviceSynchronize();


    // hipFree(cu_pm);
    // hipFree(cu_s1);
    end = clock();
    printf("\nExecuted time is: %f ms. \n\n", ((double)(end - begin) / CLOCKS_PER_SEC) * 1000);
    
    cleanup_destroy(list);

    end = clock();
    // fclose(fp);
    return 0;
}


__global__ void initilaizeTree(pm_t * cu_pm)
{
    if(pm_init_gpu(cu_pm) == -1)
    {
        return;
    }
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"Hello", 5) == -1){
        return;
    }
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"ABC153541278920", 15) == -1){
        return;
    }    
 
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"2222222222222222222222222222222222222222222222222222222222222222222222222222222222222222222222222222222222222222222", 115) == -1){
        return;
    }    
 
    // print_pm(cu_pm);    
 
    if (pm_makeFSM_gpu(cu_pm) == -1){
        return;
    }
}

__global__ void search_patterns(pm_t *pm,  char *s, int lenOfS, slist_t *list, int offset)
{
    pm_fsm_search<<< 1,1>>>(pm->zerostate, (unsigned char *)(&s[ offset + threadIdx.x]), lenOfS-threadIdx.x, offset + threadIdx.x);
}

void cleanup_destroy( slist_t *list ){
    slist_destroy(list, SLIST_FREE_DATA);
    free(list);
    // pm_destroy(pm);
}

__device__ void print_pm(pm_t *pm)
{
    if (!pm)
    {
        return;
    }
    printf("state(id, fail state id)\n");
    printf("(root)--------");
    int *b = (int *)malloc(sizeof(int) * 100);
    int i;
    for (i = 0; i < 100; i++)
    {
        b[i] = 0;
    }
    print_state(pm->zerostate, 1, b, 0);
    printf("\n");
    free(b);
}

__device__ void print_state(pm_state_t *state, int tabs, int *is_need, int is_get)
{
    slist_node_t *node = slist_head(state->_transitions);
    if (!node)
    {
        printf("\n");
        print_tabs(tabs, is_need);
        return;
    }
    int use_tabs = 0;
    while (node)
    {
        pm_labeled_edge_t *edge = (pm_labeled_edge_t *)slist_data(node);
        int is_out_state = slist_head(edge->state->output) ? 0 : -1;
        if (use_tabs != 0)
        {
            print_tabs(tabs, is_need);
            printf("%c---", edge->label);
        }

        else
        {
            printf("--|%c---", edge->label);
            use_tabs++;
        }

        int id = edge->state->id;
        int fail_id = edge->state->fail ? edge->state->fail->id : 0;
        is_out_state == 0 ? printf("(") : printf("-");
        printf("(");
        if (id < 10)
            printf(" ");
        printf("%d,", id);
        if (fail_id < 10)
            printf(" ");
        printf("%d)", fail_id);
        is_out_state == 0 ? printf(")") : printf("-");
        node = slist_next(node);
        if (node)
        {
            is_need[state->depth] = 1;
        }

        else
        {
            is_need[state->depth] = 0;
        }
        print_state(edge->state, tabs + 1, is_need, is_get);
    }
    hipDeviceSynchronize();
}

__device__ void print_tabs(int tabs, int *is_need)
{
    int i;
    for (i = 0; i < tabs; i++)
    {
        printf("\t\t\t\t");
        if (is_need[i] == 1)
        {
            printf("|");
        }
    }
}