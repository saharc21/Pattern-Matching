#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <pthread.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "slist.c"
#include "pattern_matching_aho.c"


__device__ void print_pm(pm_t *pm);
__device__ void print_state(pm_state_t *state, int tabs, int *is_need, int is_get);
__device__ void print_tabs(int tabs, int *is_need);
__global__ void search_patterns(pm_t *pm,  char *s,int lenOfS, slist_t *list);
__global__ void initilaizeTree(pm_t * cu_pm);
void cleanup_destroy(slist_t *list);

char s1[PM_CHARACTERS];
__device__ int NUM_OF_THREADS;

int main(int argc, char *argv[]) // ********************* Aho - parallel ***********************
{
    FILE *fp;
    fp = fopen("/home/jceproject/Desktop/Final Project/Pattern-Matching/chars_test.txt", "r");
    fgets(s1, PM_CHARACTERS, (FILE *)fp);
    int numOfChars = strlen(strtok(s1, "\0"));
    
    // NUM_OF_THREADS = numOfChars - 113;
    NUM_OF_THREADS = 10;
    
    // pthread_t *tids = (pthread_t *)malloc(sizeof(pthread_t) * NUM_OF_THREADS);
    
    // pm_t *pm = (pm_t *)malloc(sizeof(pm_t));
    pm_t * cu_pm;
    hipMallocManaged(&cu_pm, sizeof(pm_t));
    
    if (!cu_pm)
    {
        return -1;
    }
    
    // if (pm_init(pm) == -1)
    // {
    //     free(pm);
    //     return -1;
    // }

    initilaizeTree<<<1,1>>>(cu_pm);
    char * cu_s1;
    hipMallocManaged(&cu_s1, (numOfChars+1)*sizeof(char));
    hipMemcpy(cu_s1, s1,numOfChars,hipMemcpyHostToDevice);

    
    clock_t begin;
    clock_t end;
    begin = clock();
    
    // hipMemcpy(cu_pm, pm,sizeof(*pm),hipMemcpyHostToDevice);

    // hipMallocManaged(&(cu_pm->zerostate), sizeof(pm->zerostate));
    // hipMemcpy(cu_pm->zerostate, pm->zerostate,sizeof(pm->zerostate),hipMemcpyHostToDevice);
    
    // hipMallocManaged(&(cu_pm->zerostate->_transitions), sizeof(pm->zerostate->_transitions));
    // hipMemcpy(cu_pm->zerostate->_transitions, pm->zerostate->_transitions,sizeof(pm->zerostate->_transitions),hipMemcpyHostToDevice);



    slist_t *list = NULL;
    int len = strlen(s1);


    // printf("gpu tree: \n");
    // print_pm(cu_pm);
    search_patterns<<< 1, NUM_OF_THREADS>>>(&(*cu_pm), cu_s1, len, list);

    printf("in main\n");
    hipDeviceSynchronize();


    // hipFree(cu_pm);
    // hipFree(cu_s1);
    
    cleanup_destroy(list);

    end = clock();
    // fclose(fp);
    printf("\nExecuted time is: %f ms. \n\n", ((double)(end - begin) / CLOCKS_PER_SEC) * 1000);
    return 0;
}


__global__ void initilaizeTree(pm_t * cu_pm)
{
    if(pm_init_gpu(cu_pm) == -1)
    {
        return;
    }
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"hello", 5) == -1){
        return;
    }    
    if (pm_addstring_gpu(cu_pm, (unsigned char *)"GPU", 3) == -1){
        return;
    }
    if (pm_makeFSM_gpu(cu_pm) == -1){
        return;
    }
    print_pm(cu_pm);    
}

__global__ void search_patterns(pm_t *pm,  char *s, int lenOfS, slist_t *list)
{
    pm_fsm_search<<< 1,1>>>(pm->zerostate, (unsigned char *)(&s[threadIdx.x]), lenOfS-threadIdx.x, threadIdx.x);
}

void cleanup_destroy( slist_t *list ){
    slist_destroy(list, SLIST_FREE_DATA);
    free(list);
    // pm_destroy(pm);
}

__device__ void print_pm(pm_t *pm)
{
    if (!pm)
    {
        return;
    }
    printf("state(id, fail state id)\n");
    printf("(root)--------");
    int *b = (int *)malloc(sizeof(int) * 100);
    int i;
    for (i = 0; i < 100; i++)
    {
        b[i] = 0;
    }
    print_state(pm->zerostate, 1, b, 0);
    printf("\n");
    free(b);
}

__device__ void print_state(pm_state_t *state, int tabs, int *is_need, int is_get)
{
    slist_node_t *node = slist_head(state->_transitions);
    if (!node)
    {
        printf("\n");
        print_tabs(tabs, is_need);
        return;
    }
    int use_tabs = 0;
    while (node)
    {
        pm_labeled_edge_t *edge = (pm_labeled_edge_t *)slist_data(node);
        int is_out_state = slist_head(edge->state->output) ? 0 : -1;
        if (use_tabs != 0)
        {
            print_tabs(tabs, is_need);
            printf("%c---", edge->label);
        }

        else
        {
            printf("--|%c---", edge->label);
            use_tabs++;
        }

        int id = edge->state->id;
        int fail_id = edge->state->fail ? edge->state->fail->id : 0;
        is_out_state == 0 ? printf("(") : printf("-");
        printf("(");
        if (id < 10)
            printf(" ");
        printf("%d,", id);
        if (fail_id < 10)
            printf(" ");
        printf("%d)", fail_id);
        is_out_state == 0 ? printf(")") : printf("-");
        node = slist_next(node);
        if (node)
        {
            is_need[state->depth] = 1;
        }

        else
        {
            is_need[state->depth] = 0;
        }
        print_state(edge->state, tabs + 1, is_need, is_get);
    }
    // print_tabs(tabs, is_need);
    hipDeviceSynchronize();
}

__device__ void print_tabs(int tabs, int *is_need)
{
    int i;
    for (i = 0; i < tabs; i++)
    {
        printf("\t\t\t\t");
        if (is_need[i] == 1)
        {
            printf("|");
        }
    }
}