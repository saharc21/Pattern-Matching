#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <pthread.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "slist.c"
#include "pattern_matching_dfa.c"


__device__ void print_pm(pm_t *pm);
__device__ void print_state(pm_state_t *state, int tabs, int *is_need, int is_get);
__device__ void print_tabs(int tabs, int *is_need);
__global__ void search_patterns(pm_t **pm, unsigned char *s, int lenOfS);
__global__ void initilaizeTrees_dfa(pm_t ** cu_pms, pattern_details** patternArray, int numOfTrees);
__global__ void allocateStaticPacket(char *d, int n);
__global__ void f ();
void cleanup_destroy(slist_t *list);

char s1[PM_CHARACTERS+1];
int MAX_THEADS_FOR_EPOCH;
int NUM_OF_THREADS;

unsigned char * cu_s1;

int main(int argc, char *argv[]) // ********************* DFA - parallel ***********************
{
    FILE *fp;
    fp = fopen("/home/jceproject/Desktop/Final Project/Pattern-Matching/chars_stream.txt", "r");
    fgets(s1, PM_CHARACTERS+1, (FILE *)fp);
    int numOfChars = strlen(strtok(s1, "\0"));
    int numOfTrees = 112;


    pm_t** cu_pms;
    hipMallocManaged(&cu_pms, numOfTrees*sizeof(pm_t*));//cu_pms
    
    pattern_details** patternArray;
    hipMallocManaged(&patternArray, numOfTrees*sizeof(pattern_details*));//patternArray
    
    int i;
    for(i = 0; i< numOfTrees; i++){
        hipMallocManaged(&(patternArray[i]), sizeof(pattern_details));//patternArray[i]
    }
    
    
    patternArray[0]->len = 479;
    hipMallocManaged(&(patternArray[0]->string), patternArray[0]->len);
    hipMemcpy(patternArray[0]->string,"DR5HY@yLcG*6GD7sXf1paMwfLiD&Gdub98QW0FzqpWjEHxMQZ0kJoZo#hJVJ9QGD3VdtIadTp4me*sfAzT3NjL0$4L2FC6NzmDd6mhtyW04o@*qO#le31L^MO$X#CpFYxHSegmPDjiCf8R^cmjtARILA*Z^e!n&pPEC&WKKNHO7I4P6XPA0&#75RJ7ghWsHXz^r3vSh@OzJK#jqaZBvz9x4EF*$hlfcrykrKdLgio7AaxLiuDD805i@4&s4WWusG7fsf5iNllmMrVkb7RtzSU@fZo1bLVaceLA#CkuWSpvaD0b#ZGwkcTesehS5UwKUObzILks#sdUJDBdadWpgP&lanjk^CbqjbumPvz2chvCa&Sc120z5p60TEDBc9esY3vbtq8w5j^M0K$$UYZB2NMAUjJ*kK^1SoX5f03yxAh0#!R$92Ui&88zhwxsnq2evaygzksj##w1Sx^jvknA&Vyba^Dt1j8CM", patternArray[0]->len, hipMemcpyHostToDevice);
    patternArray[1]->len = 378;
    hipMallocManaged(&(patternArray[1]->string), patternArray[1]->len);
    hipMemcpy(patternArray[1]->string,"#^1wds5KDaplo&!mEFt7Xak&SFpa0fNP8xZwdmRXS7tZ2!B9P4na&yHXT*JAsuGUmeYypacsvkxnkUDgC*GWmUBf5WKT8E2k!UF@pK3g86ansmrmOfspzeSGjJmjfmg9#*pOYs@b^cGZLycrtv@*wrxCgeVvb&4v3mZwRwDgdEPZMQX0lyANM23HL3oWm#dm2nS6A!OAGq$QjoA9*DK&*ffQ2&wZxJf06Vxck1om!ccKe7qpqZk5iJECurt!3s6EpTPdpgxeKAZLlR4Bt1V#5ZBfX4tW0^3I0zlJvomrNOiOKi!vK^8lX36JkuJCzhrvAKk#aC^RtQAtS$qK^A*30I^$Ed3KZ82rgwG7z4mswABIzOH*3bm!6C@dCx", patternArray[1]->len, hipMemcpyHostToDevice);
    patternArray[2]->len = 124;
    hipMallocManaged(&(patternArray[2]->string), patternArray[2]->len);
    hipMemcpy(patternArray[2]->string,"lKEX*VA$uFqJ2FfJE5*EI!MU5vMg!KGq9xP$fAshHrIVXgQfL45&1tT&TOUcTcKUYGoAgTEIrl6NiL8n&DPIvmomUpBp2sHM#*g1V2&xsYFOOznYQwzoqnBp2w*P", patternArray[2]->len, hipMemcpyHostToDevice);
    patternArray[3]->len = 632;
    hipMallocManaged(&(patternArray[3]->string), patternArray[3]->len);
    hipMemcpy(patternArray[3]->string,"dA1sI0NGxYzGPWQreWrptknA9SMf44Bfl9hXDAV!TvWSGB!1e66$2wG6tx&Re*LoEBwJ!dUL@qOJuJ2kffeazN1Mhcvuc4$J7Xjox$1fAwSW6ziIvlv7PstENidAXl7DnlR4W7*yCjz$XhPvmWLflmvohg!XxTI*c#Ra5QRL2HKmc&PCzjQM*wzjSbbxLqC*G$Qe^nFIM8nEzfV4keXikucg$Z1GCW04eBYTWp*HCwrHHGqtC*XmRsPx@iGMQg5yGkeXQMiiQJUyLnp1f3PC4evR@^zijZ$H&XkZ7XuTUTcVV8vkIUpzoTRjYw$C1arvd&Y!&rFY35*RFDj!bv&&yl*sN4W3ALkDGXudUKRlPFYy!D89kXQmwkyHjO*&RAv#eJ2vBN7#*NMN*eY0bD$xQ0Rn7ywTUcch^fiU@3s$$2oGH2zjVLKXhHFVeRb7X9al9T^vwldPKFJDgqJgVky!QgWQ5g#*^lsEMykUV6o#h2UPhoy3@rqscqeBJ!GJwEJWj$Li^bynEWhtTsojwCiaB29gU&thp0O5$PI1Lq9UzpLTc8ZZ1$OgJSN11xJzpjC1&mNh7T@wD8j&M&Rjk^TXZckBHT9fVmS1!F0GHslpaBTZZ#DD6p@!CkDk", patternArray[3]->len, hipMemcpyHostToDevice);
    patternArray[4]->len = 212;
    hipMallocManaged(&(patternArray[4]->string), patternArray[4]->len);
    hipMemcpy(patternArray[4]->string,"gGoojjNNZnj09JmxraEvWyaO9raPqdABd0iZSqoMVyPtyIfpLolELRVtyAjYyIE4x64spFYW6jXCE5CIRMdxaO0YfzlvkFYHwM1gSUbOqxuXkIYg9WFuQ6MG!VFoyoJ1xk0Y96mSP7eUhOmZeU9rzItkRzRVfvlKSvmxYwX7yoOIPVcxhL46ufZnn199V225aeKi8qXrCoUxiGELsNpV", patternArray[4]->len, hipMemcpyHostToDevice);
    patternArray[5]->len = 729;
    hipMallocManaged(&(patternArray[5]->string), patternArray[5]->len);
    hipMemcpy(patternArray[5]->string,"E4yE1Zj7vkE2mNzp8Ikimo2lIEAFFyKZEsz7N7Sue629yw8RJ3zvHxWleH9KJqt0t9GTfL75eZqayVQOzHagHwLdacJHjaZ2ul4zHCb1af7CPpsSp6wstt8xBhqAaSydUCJcoFyetcxEh0zM1MiMDKEyDHmq4OhvSMFbI3SbgGoZ8SiMvCcWX2b0YsQVgF65LoEEKIL71xye0pBNxD4bNoQgvFTcNxK33mGB37oNekERK5qUOvO9xpKlWIaTDtkOTTnXL6WjK0WKbONOCw8x1kCKwk12aVUgM27L3kLTIdo0s6VUVAAGeqYFNRYzyn2F7qJSpEiygYaNdLkiJdLcf0QguBhhIOfTKh4henpss2Z2wBStst1lZvrZL6R5OjwMEFsUEsUjS8z0nOoFEAl1TOY4P9elaAPQ7ornOFAUybAVOZEmlu2atvLJVvv1VCsIEmwnezgyULFG886hbVRO5Z9uxWti2G8FmcI09JQkuXZ3vAyyEUVmLSJhc1tKMVpv8dRKq61ROSfs6f1qhepGxoJZQTkMRMaSii71JW6Fv0XzSSrsYEylWgkRNS1JUkB3Qe8YeryJ9ubX5I5VCd0JoStAY704klkXDUE2bKNMFALcFNjVDUdjpeu7q1TslRLFbpSrdfionpmmFyIzopE4CAYkz2jlablPHLUeBMjGgiBgbYOulSIKSuwx7rlvFCbhyTsibHtZxEY6TQEOkTWSPMXqp", patternArray[5]->len, hipMemcpyHostToDevice);
    patternArray[6]->len = 311;
    hipMallocManaged(&(patternArray[6]->string), patternArray[6]->len);
    hipMemcpy(patternArray[6]->string,"YBx1TflInCLoTvSMvcgYv2Xd916G1nwiVqdpgmFdvFykLJPJRtSfVD2oiGccXNeSuTq1Pqt25hfFtIKGJDO8SVQdI5W24qLsUUJIgo31tAjbg4D6HFngMcApiewrd5CAWW41an8N5h6ZXtYM5pLytzkWddE4esqvEKHM9eRBdVpkbC0pXmVYeNCoSSjR7pwXxfSw4nk7Q0LSAdbbMgOIijc8z7wvfRLdbzuIgjVbR9USyiNm4fSYizzqc4fbQGdCx7nuNk1hxmFhS3Wpp70rP3Q7hrD4VJe3p2HlneNnpoLbJOWhJwVt7ze", patternArray[6]->len, hipMemcpyHostToDevice);
    patternArray[7]->len = 657;
    hipMallocManaged(&(patternArray[7]->string), patternArray[7]->len);
    hipMemcpy(patternArray[7]->string,"lC14n8wwW12TxLorJKoO5PLRqSuafVm5MFC2OEzhvY5XD07nHYwGdzfDQ8XmUzUmyKNPjBp62AxJAJ97V9WPYAKHJDmKZ4AvSH1n2Ho651F9WgqbXzj3Abdr3m5wtDY8cb3kS3lRYcmNepUlOSuHw2Gy3EtPFLHVLyF2Fjls0mK6N2kxOR023jQTWwpqdHUN6MnTLn61Mx2c9sDrwSmvxaPcsGXhA6pnnPczdJhts6XvdIXmKaQzYpaC3niN82tSjwtR0VSsOeAE3WHZvjTxAduYagif23clk23QIdoenFqpxnqBLhFD9ShEfSuot28Ta0hcaIyxKuMi9KjD1f8HRM7mZ7zwQ3KaeSLoi7ovgWcUXMFkkCQsZ6rFLZlGRgBEFz2rfwXOmU0rGA94DIejZ6zy8VtI6D2nY13fsyPPGTpx80OwcCQhDEoJ83bEd1zlzwK9F15RDupRAmpy6eMz7BRosGy3rqgtauypdqbQ7vxWJ62X3dALkAwWgUu1WQNPRQpax2cusZ12igIJxRqd2MIMJw93C2BAU4eXphcFimgGLWlS3mUAtDP8ZUHqGjkACC4AQ6Fy3CigMKpLWAn7wPytFDkTXmVb0EeO0Mc6SEOOSWXbXukBT37kPMJlDcFL0pQBDqsuUp5Tny6EU", patternArray[7]->len, hipMemcpyHostToDevice);
    patternArray[8]->len = 113;
    hipMallocManaged(&(patternArray[8]->string), patternArray[8]->len);
    hipMemcpy(patternArray[8]->string,"mjXnO5lUQKGB1RnpYMXq72alSUcZLthkcvCYuwmeTfThqjVbe0PzN6hId9a6u9DGxL37pbPcyBycDYWZmhwPF6CdQzyNxlAhLyjU0FG9jYICzxsGt", patternArray[8]->len, hipMemcpyHostToDevice);
    patternArray[9]->len = 777;
    hipMallocManaged(&(patternArray[9]->string), patternArray[9]->len);
    hipMemcpy(patternArray[9]->string,"kDR1XpjpFrgtED8y24YD76zgJz5hIAEfRJylpD7wO8oHE0xyjPUMTcqEabuZtvnUNgfZZjclcPlwOzJALTQQ1KzlGzfrGxKnbxRYiCw0IXgfdgAINS9gaNTVVT2AYIfxG5oiTqSFGNe8mx2inlzPHJSdZLx7Gh0Rmncf5MESgPxdkVKHxS0dGHEx9WIbKiimQMy5LHl2RwRTMWybSy8X1PYiXcAGw1x9HQ7QjavVkjBBLt8GnbeulhV0dN2wTO9gnoIjhulLzRltiHx57vafQM4fOalysOUqfhUppN494uD482cWkuNq08bAaNjaNq19gL9hI3AJNXzFl7KH1f6h3xd0qGuunOQv5N25U9X676qRm3hVlkgra8EVKRMW2vbCO9SE55py9k7Ehg6lrnPy0WayZfJly2ZlgwtFqVFR7dBXYklRZcru3RQdVDFOvObNJmJPx0A82LC1Q0bqpAzuSQQ2mOWKhg6bEF0q2tWSB3d0I9m8I97MoCKDWDfIHBrpHLW1GbDJJdZCePakRSn6ityD0reUreEv3Nk3oKGY3mhVRqvwq1K374D80HBudTsYNP6xa1yJtANEus3YnqI38AQ9eu46q1xfmYfY28V9mvOviAnj3G9nGRkdIzO6CkNIPWZ7T0TTlEmftZKe1MIXZYW2AnIXaSn5YCmzT6rYqIpT0lO4TGTX8Nu2Aj8GDbhAj0Zs0cnxS5b3FSylzp8MHuJsdyNxgMGTeNtGloWQwvfhHdCsj9rEbchclaBriYZtx7rzaxbIB", patternArray[9]->len, hipMemcpyHostToDevice);


    patternArray[10]->len = 17;
    hipMallocManaged(&(patternArray[10]->string), patternArray[10]->len);
    hipMemcpy(patternArray[10]->string,"check1=1234567890", patternArray[10]->len, hipMemcpyHostToDevice);

    patternArray[11]->len = 17;
    hipMallocManaged(&(patternArray[11]->string), patternArray[11]->len);
    hipMemcpy(patternArray[11]->string,"check2=1234567890", patternArray[11]->len, hipMemcpyHostToDevice);

    patternArray[12]->len = 17;
    hipMallocManaged(&(patternArray[12]->string), patternArray[12]->len);
    hipMemcpy(patternArray[12]->string,"check3=1234567890", patternArray[12]->len, hipMemcpyHostToDevice);

    patternArray[13]->len = 17;
    hipMallocManaged(&(patternArray[13]->string), patternArray[13]->len);
    hipMemcpy(patternArray[13]->string,"check4=1234567890", patternArray[13]->len, hipMemcpyHostToDevice);

    patternArray[14]->len = 17;
    hipMallocManaged(&(patternArray[14]->string), patternArray[14]->len);
    hipMemcpy(patternArray[14]->string,"check5=1234567890", patternArray[14]->len, hipMemcpyHostToDevice);

    patternArray[15]->len = 17;
    hipMallocManaged(&(patternArray[15]->string), patternArray[15]->len);
    hipMemcpy(patternArray[15]->string,"check6=1234567890", patternArray[15]->len, hipMemcpyHostToDevice);

    patternArray[16]->len = 17;
    hipMallocManaged(&(patternArray[16]->string), patternArray[16]->len);
    hipMemcpy(patternArray[16]->string,"check7=1234567890", patternArray[16]->len, hipMemcpyHostToDevice);

    patternArray[17]->len = 17;
    hipMallocManaged(&(patternArray[17]->string), patternArray[17]->len);
    hipMemcpy(patternArray[17]->string,"check8=1234567890", patternArray[17]->len, hipMemcpyHostToDevice);

    patternArray[18]->len = 17;
    hipMallocManaged(&(patternArray[18]->string), patternArray[18]->len);
    hipMemcpy(patternArray[18]->string,"check9=1234567890", patternArray[18]->len, hipMemcpyHostToDevice);

    patternArray[19]->len = 18;
    hipMallocManaged(&(patternArray[19]->string), patternArray[19]->len);
    hipMemcpy(patternArray[19]->string,"check10=1234567890", patternArray[19]->len, hipMemcpyHostToDevice);

    patternArray[20]->len = 18;
    hipMallocManaged(&(patternArray[20]->string), patternArray[20]->len);
    hipMemcpy(patternArray[20]->string,"check11=1234567890", patternArray[20]->len, hipMemcpyHostToDevice);

    patternArray[21]->len = 18;
    hipMallocManaged(&(patternArray[21]->string), patternArray[21]->len);
    hipMemcpy(patternArray[21]->string,"check12=1234567890", patternArray[21]->len, hipMemcpyHostToDevice);

    patternArray[22]->len = 18;
    hipMallocManaged(&(patternArray[22]->string), patternArray[22]->len);
    hipMemcpy(patternArray[22]->string,"check13=1234567890", patternArray[22]->len, hipMemcpyHostToDevice);

    patternArray[23]->len = 18;
    hipMallocManaged(&(patternArray[23]->string), patternArray[23]->len);
    hipMemcpy(patternArray[23]->string,"check14=1234567890", patternArray[23]->len, hipMemcpyHostToDevice);

    patternArray[24]->len = 18;
    hipMallocManaged(&(patternArray[24]->string), patternArray[24]->len);
    hipMemcpy(patternArray[24]->string,"check15=1234567890", patternArray[24]->len, hipMemcpyHostToDevice);

    patternArray[25]->len = 18;
    hipMallocManaged(&(patternArray[25]->string), patternArray[25]->len);
    hipMemcpy(patternArray[25]->string,"check16=1234567890", patternArray[25]->len, hipMemcpyHostToDevice);

    patternArray[26]->len = 18;
    hipMallocManaged(&(patternArray[26]->string), patternArray[26]->len);
    hipMemcpy(patternArray[26]->string,"check17=1234567890", patternArray[26]->len, hipMemcpyHostToDevice);

    patternArray[27]->len = 18;
    hipMallocManaged(&(patternArray[27]->string), patternArray[27]->len);
    hipMemcpy(patternArray[27]->string,"check18=1234567890", patternArray[27]->len, hipMemcpyHostToDevice);

    patternArray[28]->len = 18;
    hipMallocManaged(&(patternArray[28]->string), patternArray[28]->len);
    hipMemcpy(patternArray[28]->string,"check19=1234567890", patternArray[28]->len, hipMemcpyHostToDevice);

    patternArray[29]->len = 18;
    hipMallocManaged(&(patternArray[29]->string), patternArray[29]->len);
    hipMemcpy(patternArray[29]->string,"check20=1234567890", patternArray[29]->len, hipMemcpyHostToDevice);
    
    patternArray[30]->len = 18;
    hipMallocManaged(&(patternArray[30]->string), patternArray[30]->len);
    hipMemcpy(patternArray[30]->string,"check21=1234567890", patternArray[30]->len, hipMemcpyHostToDevice);

    patternArray[31]->len = 18;
    hipMallocManaged(&(patternArray[31]->string), patternArray[31]->len);
    hipMemcpy(patternArray[31]->string,"check22=1234567890", patternArray[31]->len, hipMemcpyHostToDevice);

    patternArray[32]->len = 18;
    hipMallocManaged(&(patternArray[32]->string), patternArray[32]->len);
    hipMemcpy(patternArray[32]->string,"check23=1234567890", patternArray[32]->len, hipMemcpyHostToDevice);

    patternArray[33]->len = 18;
    hipMallocManaged(&(patternArray[33]->string), patternArray[33]->len);
    hipMemcpy(patternArray[33]->string,"check24=1234567890", patternArray[33]->len, hipMemcpyHostToDevice);

    patternArray[34]->len = 19;
    hipMallocManaged(&(patternArray[34]->string), patternArray[34]->len);
    hipMemcpy(patternArray[34]->string,"check103=1234567890", patternArray[34]->len, hipMemcpyHostToDevice);

    patternArray[35]->len = 18;
    hipMallocManaged(&(patternArray[35]->string), patternArray[35]->len);
    hipMemcpy(patternArray[35]->string,"check25=1234567890", patternArray[35]->len, hipMemcpyHostToDevice);

    patternArray[36]->len = 18;
    hipMallocManaged(&(patternArray[36]->string), patternArray[36]->len);
    hipMemcpy(patternArray[36]->string,"check26=1234567890", patternArray[36]->len, hipMemcpyHostToDevice);

    patternArray[37]->len = 18;
    hipMallocManaged(&(patternArray[37]->string), patternArray[37]->len);
    hipMemcpy(patternArray[37]->string,"check27=1234567890", patternArray[37]->len, hipMemcpyHostToDevice);

    patternArray[38]->len = 18;
    hipMallocManaged(&(patternArray[38]->string), patternArray[38]->len);
    hipMemcpy(patternArray[38]->string,"check28=1234567890", patternArray[38]->len, hipMemcpyHostToDevice);

    patternArray[39]->len = 18;
    hipMallocManaged(&(patternArray[39]->string), patternArray[39]->len);
    hipMemcpy(patternArray[39]->string,"check29=1234567890", patternArray[39]->len, hipMemcpyHostToDevice);

    patternArray[40]->len = 18;
    hipMallocManaged(&(patternArray[40]->string), patternArray[40]->len);
    hipMemcpy(patternArray[40]->string,"check30=1234567890", patternArray[40]->len, hipMemcpyHostToDevice);

    patternArray[41]->len = 19;
    hipMallocManaged(&(patternArray[41]->string), patternArray[41]->len);
    hipMemcpy(patternArray[41]->string,"check101=1234567890", patternArray[41]->len, hipMemcpyHostToDevice);

    patternArray[42]->len = 18;
    hipMallocManaged(&(patternArray[42]->string), patternArray[42]->len);
    hipMemcpy(patternArray[42]->string,"check31=1234567890", patternArray[42]->len, hipMemcpyHostToDevice);

    patternArray[43]->len = 18;
    hipMallocManaged(&(patternArray[43]->string), patternArray[43]->len);
    hipMemcpy(patternArray[43]->string,"check32=1234567890", patternArray[43]->len, hipMemcpyHostToDevice);

    patternArray[44]->len = 18;
    hipMallocManaged(&(patternArray[44]->string), patternArray[44]->len);
    hipMemcpy(patternArray[44]->string,"check33=1234567890", patternArray[44]->len, hipMemcpyHostToDevice);

    patternArray[45]->len = 18;
    hipMallocManaged(&(patternArray[45]->string), patternArray[45]->len);
    hipMemcpy(patternArray[45]->string,"check34=1234567890", patternArray[45]->len, hipMemcpyHostToDevice);

    patternArray[46]->len = 19;
    hipMallocManaged(&(patternArray[46]->string), patternArray[46]->len);
    hipMemcpy(patternArray[46]->string,"check102=1234567890", patternArray[46]->len, hipMemcpyHostToDevice);

    patternArray[47]->len = 18;
    hipMallocManaged(&(patternArray[47]->string), patternArray[47]->len);
    hipMemcpy(patternArray[47]->string,"check35=1234567890", patternArray[47]->len, hipMemcpyHostToDevice);

    patternArray[48]->len = 18;
    hipMallocManaged(&(patternArray[48]->string), patternArray[48]->len);
    hipMemcpy(patternArray[48]->string,"check36=1234567890", patternArray[48]->len, hipMemcpyHostToDevice);

    patternArray[49]->len = 18;
    hipMallocManaged(&(patternArray[49]->string), patternArray[49]->len);
    hipMemcpy(patternArray[49]->string,"check37=1234567890", patternArray[49]->len, hipMemcpyHostToDevice);

    patternArray[50]->len = 18;
    hipMallocManaged(&(patternArray[50]->string), patternArray[50]->len);
    hipMemcpy(patternArray[50]->string,"check38=1234567890", patternArray[50]->len, hipMemcpyHostToDevice);

    patternArray[51]->len = 18;
    hipMallocManaged(&(patternArray[51]->string), patternArray[51]->len);
    hipMemcpy(patternArray[51]->string,"check39=1234567890", patternArray[51]->len, hipMemcpyHostToDevice);

    patternArray[52]->len = 18;
    hipMallocManaged(&(patternArray[52]->string), patternArray[52]->len);
    hipMemcpy(patternArray[52]->string,"check40=1234567890", patternArray[52]->len, hipMemcpyHostToDevice);

    patternArray[53]->len = 18;
    hipMallocManaged(&(patternArray[53]->string), patternArray[53]->len);
    hipMemcpy(patternArray[53]->string,"check41=1234567890", patternArray[53]->len, hipMemcpyHostToDevice);

    patternArray[54]->len = 18;
    hipMallocManaged(&(patternArray[54]->string), patternArray[54]->len);
    hipMemcpy(patternArray[54]->string,"check42=1234567890", patternArray[54]->len, hipMemcpyHostToDevice);

    patternArray[55]->len = 18;
    hipMallocManaged(&(patternArray[55]->string), patternArray[55]->len);
    hipMemcpy(patternArray[55]->string,"check43=1234567890", patternArray[55]->len, hipMemcpyHostToDevice);

    patternArray[56]->len = 18;
    hipMallocManaged(&(patternArray[56]->string), patternArray[56]->len);
    hipMemcpy(patternArray[56]->string,"check44=1234567890", patternArray[56]->len, hipMemcpyHostToDevice);

    patternArray[57]->len = 18;
    hipMallocManaged(&(patternArray[57]->string), patternArray[57]->len);
    hipMemcpy(patternArray[57]->string,"check45=1234567890", patternArray[57]->len, hipMemcpyHostToDevice);

    patternArray[58]->len = 18;
    hipMallocManaged(&(patternArray[58]->string), patternArray[58]->len);
    hipMemcpy(patternArray[58]->string,"check46=1234567890", patternArray[58]->len, hipMemcpyHostToDevice);

    patternArray[59]->len = 18;
    hipMallocManaged(&(patternArray[59]->string), patternArray[59]->len);
    hipMemcpy(patternArray[59]->string,"check47=1234567890", patternArray[59]->len, hipMemcpyHostToDevice);

    patternArray[60]->len = 18;
    hipMallocManaged(&(patternArray[60]->string), patternArray[60]->len);
    hipMemcpy(patternArray[60]->string,"check48=1234567890", patternArray[60]->len, hipMemcpyHostToDevice);

    patternArray[61]->len = 18;
    hipMallocManaged(&(patternArray[61]->string), patternArray[61]->len);
    hipMemcpy(patternArray[61]->string,"check49=1234567890", patternArray[61]->len, hipMemcpyHostToDevice);

    patternArray[62]->len = 18;
    hipMallocManaged(&(patternArray[62]->string), patternArray[62]->len);
    hipMemcpy(patternArray[62]->string,"check50=1234567890", patternArray[62]->len, hipMemcpyHostToDevice);

    patternArray[63]->len = 18;
    hipMallocManaged(&(patternArray[63]->string), patternArray[63]->len);
    hipMemcpy(patternArray[63]->string,"check51=1234567890", patternArray[63]->len, hipMemcpyHostToDevice);

    patternArray[64]->len = 18;
    hipMallocManaged(&(patternArray[64]->string), patternArray[64]->len);
    hipMemcpy(patternArray[64]->string,"check52=1234567890", patternArray[64]->len, hipMemcpyHostToDevice);

    patternArray[65]->len = 18;
    hipMallocManaged(&(patternArray[65]->string), patternArray[65]->len);
    hipMemcpy(patternArray[65]->string,"check53=1234567890", patternArray[65]->len, hipMemcpyHostToDevice);

    patternArray[66]->len = 18;
    hipMallocManaged(&(patternArray[66]->string), patternArray[66]->len);
    hipMemcpy(patternArray[66]->string,"check54=1234567890", patternArray[66]->len, hipMemcpyHostToDevice);

    patternArray[67]->len = 18;
    hipMallocManaged(&(patternArray[67]->string), patternArray[67]->len);
    hipMemcpy(patternArray[67]->string,"check55=1234567890", patternArray[67]->len, hipMemcpyHostToDevice);

    patternArray[68]->len = 18;
    hipMallocManaged(&(patternArray[68]->string), patternArray[68]->len);
    hipMemcpy(patternArray[68]->string,"check56=1234567890", patternArray[68]->len, hipMemcpyHostToDevice);

    patternArray[69]->len = 18;
    hipMallocManaged(&(patternArray[69]->string), patternArray[69]->len);
    hipMemcpy(patternArray[69]->string,"check57=1234567890", patternArray[69]->len, hipMemcpyHostToDevice);

    patternArray[70]->len = 18;
    hipMallocManaged(&(patternArray[70]->string), patternArray[70]->len);
    hipMemcpy(patternArray[70]->string,"check58=1234567890", patternArray[70]->len, hipMemcpyHostToDevice);

    patternArray[71]->len = 18;
    hipMallocManaged(&(patternArray[71]->string), patternArray[71]->len);
    hipMemcpy(patternArray[71]->string,"check59=1234567890", patternArray[71]->len, hipMemcpyHostToDevice);

    patternArray[72]->len = 18;
    hipMallocManaged(&(patternArray[72]->string), patternArray[72]->len);
    hipMemcpy(patternArray[72]->string,"check60=1234567890", patternArray[72]->len, hipMemcpyHostToDevice);

    patternArray[73]->len = 18;
    hipMallocManaged(&(patternArray[73]->string), patternArray[73]->len);
    hipMemcpy(patternArray[73]->string,"check61=1234567890", patternArray[73]->len, hipMemcpyHostToDevice);

    patternArray[74]->len = 18;
    hipMallocManaged(&(patternArray[74]->string), patternArray[74]->len);
    hipMemcpy(patternArray[74]->string,"check62=1234567890", patternArray[74]->len, hipMemcpyHostToDevice);

    patternArray[75]->len = 18;
    hipMallocManaged(&(patternArray[75]->string), patternArray[75]->len);
    hipMemcpy(patternArray[75]->string,"check63=1234567890", patternArray[75]->len, hipMemcpyHostToDevice);

    patternArray[76]->len = 18;
    hipMallocManaged(&(patternArray[76]->string), patternArray[76]->len);
    hipMemcpy(patternArray[76]->string,"check64=1234567890", patternArray[76]->len, hipMemcpyHostToDevice);

    patternArray[77]->len = 18;
    hipMallocManaged(&(patternArray[77]->string), patternArray[77]->len);
    hipMemcpy(patternArray[77]->string,"check65=1234567890", patternArray[77]->len, hipMemcpyHostToDevice);

    patternArray[78]->len = 18;
    hipMallocManaged(&(patternArray[78]->string), patternArray[78]->len);
    hipMemcpy(patternArray[78]->string,"check66=1234567890", patternArray[78]->len, hipMemcpyHostToDevice);

    patternArray[79]->len = 18;
    hipMallocManaged(&(patternArray[79]->string), patternArray[79]->len);
    hipMemcpy(patternArray[79]->string,"check67=1234567890", patternArray[79]->len, hipMemcpyHostToDevice);

    patternArray[80]->len = 18;
    hipMallocManaged(&(patternArray[80]->string), patternArray[80]->len);
    hipMemcpy(patternArray[80]->string,"check68=1234567890", patternArray[80]->len, hipMemcpyHostToDevice);

    patternArray[81]->len = 18;
    hipMallocManaged(&(patternArray[81]->string), patternArray[81]->len);
    hipMemcpy(patternArray[81]->string,"check69=1234567890", patternArray[81]->len, hipMemcpyHostToDevice);

    patternArray[82]->len = 18;
    hipMallocManaged(&(patternArray[82]->string), patternArray[82]->len);
    hipMemcpy(patternArray[82]->string,"check70=1234567890", patternArray[82]->len, hipMemcpyHostToDevice);

    patternArray[83]->len = 18;
    hipMallocManaged(&(patternArray[83]->string), patternArray[83]->len);
    hipMemcpy(patternArray[83]->string,"check71=1234567890", patternArray[83]->len, hipMemcpyHostToDevice);

    patternArray[84]->len = 18;
    hipMallocManaged(&(patternArray[84]->string), patternArray[84]->len);
    hipMemcpy(patternArray[84]->string,"check72=1234567890", patternArray[84]->len, hipMemcpyHostToDevice);

    patternArray[85]->len = 18;
    hipMallocManaged(&(patternArray[85]->string), patternArray[85]->len);
    hipMemcpy(patternArray[85]->string,"check73=1234567890", patternArray[85]->len, hipMemcpyHostToDevice);

    patternArray[86]->len = 18;
    hipMallocManaged(&(patternArray[86]->string), patternArray[86]->len);
    hipMemcpy(patternArray[86]->string,"check74=1234567890", patternArray[86]->len, hipMemcpyHostToDevice);

    patternArray[87]->len = 18;
    hipMallocManaged(&(patternArray[87]->string), patternArray[87]->len);
    hipMemcpy(patternArray[87]->string,"check75=1234567890", patternArray[87]->len, hipMemcpyHostToDevice);

    patternArray[88]->len = 18;
    hipMallocManaged(&(patternArray[88]->string), patternArray[88]->len);
    hipMemcpy(patternArray[88]->string,"check76=1234567890", patternArray[88]->len, hipMemcpyHostToDevice);

    patternArray[89]->len = 18;
    hipMallocManaged(&(patternArray[89]->string), patternArray[89]->len);
    hipMemcpy(patternArray[89]->string,"check77=1234567890", patternArray[89]->len, hipMemcpyHostToDevice);

    patternArray[90]->len = 18;
    hipMallocManaged(&(patternArray[90]->string), patternArray[90]->len);
    hipMemcpy(patternArray[90]->string,"check78=1234567890", patternArray[90]->len, hipMemcpyHostToDevice);

    patternArray[91]->len = 18;
    hipMallocManaged(&(patternArray[91]->string), patternArray[91]->len);
    hipMemcpy(patternArray[91]->string,"check79=1234567890", patternArray[91]->len, hipMemcpyHostToDevice);

    patternArray[92]->len = 18;
    hipMallocManaged(&(patternArray[92]->string), patternArray[92]->len);
    hipMemcpy(patternArray[92]->string,"check80=1234567890", patternArray[92]->len, hipMemcpyHostToDevice);

    patternArray[93]->len = 18;
    hipMallocManaged(&(patternArray[93]->string), patternArray[93]->len);
    hipMemcpy(patternArray[93]->string,"check81=1234567890", patternArray[93]->len, hipMemcpyHostToDevice);

    patternArray[94]->len = 18;
    hipMallocManaged(&(patternArray[94]->string), patternArray[94]->len);
    hipMemcpy(patternArray[94]->string,"check82=1234567890", patternArray[94]->len, hipMemcpyHostToDevice);

    patternArray[95]->len = 18;
    hipMallocManaged(&(patternArray[95]->string), patternArray[95]->len);
    hipMemcpy(patternArray[95]->string,"check83=1234567890", patternArray[95]->len, hipMemcpyHostToDevice);

    patternArray[96]->len = 18;
    hipMallocManaged(&(patternArray[96]->string), patternArray[96]->len);
    hipMemcpy(patternArray[96]->string,"check84=1234567890", patternArray[96]->len, hipMemcpyHostToDevice);

    patternArray[97]->len = 18;
    hipMallocManaged(&(patternArray[97]->string), patternArray[97]->len);
    hipMemcpy(patternArray[97]->string,"check85=1234567890", patternArray[97]->len, hipMemcpyHostToDevice);

    patternArray[98]->len = 18;
    hipMallocManaged(&(patternArray[98]->string), patternArray[98]->len);
    hipMemcpy(patternArray[98]->string,"check86=1234567890", patternArray[98]->len, hipMemcpyHostToDevice);

    patternArray[99]->len = 18;
    hipMallocManaged(&(patternArray[99]->string), patternArray[99]->len);
    hipMemcpy(patternArray[99]->string,"check87=1234567890", patternArray[99]->len, hipMemcpyHostToDevice);

    patternArray[100]->len = 18;
    hipMallocManaged(&(patternArray[100]->string), patternArray[100]->len);
    hipMemcpy(patternArray[100]->string,"check88=1234567890", patternArray[100]->len, hipMemcpyHostToDevice);

    patternArray[101]->len = 18;
    hipMallocManaged(&(patternArray[101]->string), patternArray[101]->len);
    hipMemcpy(patternArray[101]->string,"check89=1234567890", patternArray[101]->len, hipMemcpyHostToDevice);

    patternArray[102]->len = 18;
    hipMallocManaged(&(patternArray[102]->string), patternArray[102]->len);
    hipMemcpy(patternArray[102]->string,"check90=1234567890", patternArray[102]->len, hipMemcpyHostToDevice);

    patternArray[103]->len = 18;
    hipMallocManaged(&(patternArray[103]->string), patternArray[103]->len);
    hipMemcpy(patternArray[103]->string,"check91=1234567890", patternArray[103]->len, hipMemcpyHostToDevice);

    patternArray[104]->len = 18;
    hipMallocManaged(&(patternArray[104]->string), patternArray[104]->len);
    hipMemcpy(patternArray[104]->string,"check92=1234567890", patternArray[104]->len, hipMemcpyHostToDevice);

    patternArray[105]->len = 18;
    hipMallocManaged(&(patternArray[105]->string), patternArray[105]->len);
    hipMemcpy(patternArray[105]->string,"check93=1234567890", patternArray[105]->len, hipMemcpyHostToDevice);

    patternArray[106]->len = 18;
    hipMallocManaged(&(patternArray[106]->string), patternArray[106]->len);
    hipMemcpy(patternArray[106]->string,"check94=1234567890", patternArray[106]->len, hipMemcpyHostToDevice);

    patternArray[107]->len = 18;
    hipMallocManaged(&(patternArray[107]->string), patternArray[107]->len);
    hipMemcpy(patternArray[107]->string,"check95=1234567890", patternArray[107]->len, hipMemcpyHostToDevice);

    patternArray[108]->len = 18;
    hipMallocManaged(&(patternArray[108]->string), patternArray[108]->len);
    hipMemcpy(patternArray[108]->string,"check96=1234567890", patternArray[108]->len, hipMemcpyHostToDevice);

    patternArray[109]->len = 18;
    hipMallocManaged(&(patternArray[109]->string), patternArray[109]->len);
    hipMemcpy(patternArray[109]->string,"check97=1234567890", patternArray[109]->len, hipMemcpyHostToDevice);

    patternArray[110]->len = 18;
    hipMallocManaged(&(patternArray[110]->string), patternArray[110]->len);
    hipMemcpy(patternArray[110]->string,"check98=1234567890", patternArray[110]->len, hipMemcpyHostToDevice);

    patternArray[111]->len = 7;
    hipMallocManaged(&(patternArray[111]->string), patternArray[111]->len);
    hipMemcpy(patternArray[111]->string,"KX65fbx", patternArray[111]->len, hipMemcpyHostToDevice);

    // patternArray[112]->len = 19;
    // hipMallocManaged(&(patternArray[112]->string), patternArray[112]->len);
    // hipMemcpy(patternArray[112]->string,"check100=1234567890", patternArray[112]->len, hipMemcpyHostToDevice);

    

    if (!cu_pms)
    {
        return -1;
    }
    
    // if (pm_init(pm) == -1)
    // {
    //     free(pm);
    //     return -1;
    // }

    // for(int i=0; i<63; i++){
    //     allocateStaticPacket<<63,1024>>(s1,i);
    // }
    initilaizeTrees_dfa<<<1,1>>>(cu_pms, patternArray, numOfTrees);

    hipMalloc(&cu_s1, (numOfChars+1)*sizeof(unsigned char));//cu_s1
    hipMemcpy(cu_s1, s1,numOfChars,hipMemcpyHostToDevice);
    
    int len = strlen(s1);
    
    
    
    clock_t begin;
    clock_t end;
    
    begin = clock();
    search_patterns<<<numOfTrees,1>>>(cu_pms, cu_s1, len);
    hipDeviceSynchronize();
    end = clock();
    
    
    printf("\nExecuted time is: %f ms. \n\n", ((double)(end - begin) / CLOCKS_PER_SEC) * 1000);
    
    // hipFree(cu_pm);
    // hipFree(cu_s1);
    
    // f<<<1,1>>>();

    // cleanup_destroy(list);

    // hipFree(cu_s1);
    // for(int j=0; j<numOfTrees;j++){
    //     hipFree(patternArray[j]->string);
    // }
    // for(int j=0; j<numOfTrees;j++){
    //     hipFree(patternArray[j]);
    // }
    // hipFree(patternArray);
    // for(int j=0; j<numOfTrees;j++){
    //     hipFree(cu_pms[j]);
    // }
    // hipFree(cu_pms);
    // fclose(fp);
    return 0;
}
__global__ void f (){
    printf("in f");
}

__global__ void initilaizeTrees_dfa(pm_t ** cu_pms, pattern_details** patternArray, int numOfTrees)
{
    int i;
    for (i=0; i<numOfTrees; i++){
        cu_pms[i] = (pm_t*)malloc(sizeof(pm_t));
        if(pm_init_gpu(cu_pms[i]) == -1){
            return;
        }
        if (pm_addstring_gpu(cu_pms[i], patternArray[i]->string, patternArray[i]->len) == -1){
            return;
        }
        if (pm_makeFSM_gpu(cu_pms[i]) == -1){
            return;
        }
    }
}


__global__ void search_patterns(pm_t ** cu_pms, unsigned char *s, int lenOfS)
{   
    // __shared__ unsigned char cu_s1[40000+1];
    // int i;
    // if(threadIdx.x == 0){
    //     // cu_s1 = (unsigned char *)malloc(sizeof(unsigned char)*(lenOfS+1));
    //     for(i=0; i<40000; i++){
    //         cu_s1[i] = s[i]; 
    //     }
    // cu_s1[40000]='\0';
    // // printf("cu_s1: %s\n", cu_s1);
    // }
    // __syncthreads();
    pm_fsm_search_gpu((cu_pms[blockIdx.x])->zerostate, s, lenOfS);
}

void cleanup_destroy( slist_t *list ){
    slist_destroy(list, SLIST_FREE_DATA);
    free(list);
    // pm_destroy(pm);
}

__device__ void print_pm(pm_t *pm)
{
    if (!pm)
    {
        return;
    }
    printf("state(id, fail state id)\n");
    printf("(root)--------");
    int *b = (int *)malloc(sizeof(int) * 100);
    int i;
    for (i = 0; i < 100; i++)
    {
        b[i] = 0;
    }
    print_state(pm->zerostate, 1, b, 0);
    printf("\n");
    free(b);
}

__device__ void print_state(pm_state_t *state, int tabs, int *is_need, int is_get)
{
    slist_node_t *node = slist_head(state->_transitions);
    if (!node)
    {
        printf("\n");
        print_tabs(tabs, is_need);
        return;
    }
    int use_tabs = 0;
    while (node)
    {
        pm_labeled_edge_t *edge = (pm_labeled_edge_t *)slist_data(node);
        int is_out_state = slist_head(edge->state->output) ? 0 : -1;
        if (use_tabs != 0)
        {
            print_tabs(tabs, is_need);
            printf("%c---", edge->label);
        }

        else
        {
            printf("--|%c---", edge->label);
            use_tabs++;
        }

        int id = edge->state->id;
        int fail_id = edge->state->fail ? edge->state->fail->id : 0;
        is_out_state == 0 ? printf("(") : printf("-");
        printf("(");
        if (id < 10)
            printf(" ");
        printf("%d,", id);
        if (fail_id < 10)
            printf(" ");
        printf("%d)", fail_id);
        is_out_state == 0 ? printf(")") : printf("-");
        node = slist_next(node);
        if (node)
        {
            is_need[state->depth] = 1;
        }

        else
        {
            is_need[state->depth] = 0;
        }
        print_state(edge->state, tabs + 1, is_need, is_get);
    }
    hipDeviceSynchronize();
}

__device__ void print_tabs(int tabs, int *is_need)
{
    int i;
    for (i = 0; i < tabs; i++)
    {
        printf("\t\t\t\t");
        if (is_need[i] == 1)
        {
            printf("|");
        }
    }
}



// __global__ void allocateStaticPacket(char *d,int i)
// {
//   __shared__ char s[64000];
//     int t = blockIdx.x*i + threadIdx.x;
//     s[t] = d[t];
//   __syncthreads();
// }